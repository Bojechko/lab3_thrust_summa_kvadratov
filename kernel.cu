﻿#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

using namespace std; 
const int N = 10000000;

template<typename T>
struct square
{
    __device__ __host__ T operator ()(const T& x) const
    {
        return x * x;
    }
};




int main()
{
    thrust::device_vector<int> d(N);
    thrust::fill(d.begin(), d.end(), 3);
   

    // find the sum
    //unsigned __int64 sum = thrust::reduce(d.begin(), d.end(), 0, thrust::plus<int>());
    unsigned __int64 sum = thrust::transform_reduce(d.begin(), d.end(), square<int>(), 0, thrust::plus<int>());

    cout << "Sum is: " << sum << endl;

    return 0;
}